
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <cstdio>
#include <hipblas.h>

// 1-D row major index
#define IDX2R(i,j,w) (((i)*(w))+(j))
// 1-D column major index
#define IDX2C(i,j,h) (((j)*(h))+(i))

void printMatrix(double *h_M, int height, int width) {
	for (int i=0; i < height; i++) {
		for (int j=0; j < width; j++)
			std::cout << h_M[i * width + j] << ' ';
		std::cout << std::endl;
	}
	std::cout << std::endl;
}

void printMatrix(int *h_M, int height, int width) {
	for (int i=0; i < height; i++) {
		for (int j=0; j < width; j++)
			std::cout << h_M[i * width + j] << ' ';
		std::cout << std::endl;
	}
	std::cout << std::endl;
}

int main() {
	// cuBLAS handle
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	
	int m = 2;
	int k = 3;
	int n = 4;
	
	double *A = (double *)malloc(k*m*sizeof(double));
	double *B = (double *)malloc(k*n*sizeof(double));
	double *C = (double *)malloc(m*n*sizeof(double));
	
	double *d_A, *d_B, *d_C;
	hipMalloc(&d_A, k*m*sizeof(double));
	hipMalloc(&d_B, k*n*sizeof(double));
	hipMalloc(&d_C, m*n*sizeof(double));
	
	for (int i = 0; i < k*m; i++)	A[i] = i;
	for (int i = 0; i < k*n; i++)	B[i] = 1.0;
	
	printMatrix(A, m, k);
	printMatrix(B, k, n);
	
	hipMemcpy(d_A, A, k*m*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, k*n*sizeof(double), hipMemcpyHostToDevice);
	
	double alpha = 1.0;
	double beta = 0.0;
	hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_A, k, d_B, n, &beta, d_C, n);
	
	hipMemcpy(C, d_C, m*n*sizeof(double), hipMemcpyDeviceToHost);
	

	printMatrix(C, m, n);

	
	return 0;

}


